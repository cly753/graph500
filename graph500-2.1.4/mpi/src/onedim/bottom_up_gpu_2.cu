#include "hip/hip_runtime.h"

// 
// CUDA-aware Open MPI
// Now, the Open MPI library will automatically detect that the pointer being passed in is a CUDA device memory pointer and do the right thing. This is referred to as CUDA-aware support.
// https://www.open-mpi.org/faq/?category=runcuda
// https://devblogs.nvidia.com/parallelforall/introduction-cuda-aware-mpi/
// 

extern "C" {
#include "bottom_up.h"
}

#include <string.h>

#include "oned_csr.h"

#include "constants.h"
#include "bfs.h"
#include "print.h"

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h> // helper functions for SDK examples

// 
// !!! Assume SIZE_MUST_BE_A_POWER_OF_TWO !!!
// 

__device__ int rank_g;
__device__ int size_g;
__device__ int lgsize_g;
__device__ int64_t nlocalverts_g;

struct Context {
	int rank;
	int size;
	int lgsize;
	int64_t nlocalverts;
};

__device__ void set_context(Context context) {
	rank_g = context.rank;
	size_g = context.size;
	lgsize_g = context.lgsize;
	nlocalverts_g = context.nlocalverts;
}

// __device__ int64_t mode_size(int64_t v) {
// 	return v & ((1 << lgsize_g) - 1);
// }

// __device__ int64_t div_size(int64_t v) {
// 	return v >> lgsize_g;
// }

// __device__ int64_t mul_size(int64_t v) {
// 	return v << lgsize_g;
// }

// __device__ int vertex_owner(int64_t v) {
// 	return mode_size(v);
// }

// __device__ size_t vertex_local(int64_t v) {
// 	return div_size(v);
// }

// __device__ int64_t vertex_to_global(int r, int64_t i) {
// 	return mul_size(i) + r;
// }

#define LONG_BITS_G 64

// __device__ void set_local(int64_t v, int64_t *a) { // x / 64 --> x >> 6 ??
// 	a[vertex_local(v) / LONG_BITS_G] |= (1UL << (vertex_local(v) % LONG_BITS_G));
// }

// __device__ int test_local(int64_t v, int64_t *a) {
// 	return 0 != (a[vertex_local(v) / LONG_BITS_G] & (1UL << (vertex_local(v) % LONG_BITS_G)));
// }

// __device__ void set_global(int64_t v, int64_t *a) { // x / 64 --> x >> 6 ??
// 	a[v / LONG_BITS_G] |= (1UL << (v % LONG_BITS_G));
// }

// __device__ void set_global_atomic(int64_t v, int64_t *a) {
// 	atomicOr((unsigned long long int*)(&a[v / LONG_BITS_G]), 1UL << (v % LONG_BITS_G));
// }

// __device__ int test_global(int64_t v, int64_t *a) {
// 	return 0 != (a[v / LONG_BITS_G] & (1UL << (v % LONG_BITS_G)));
// }

#define MOD_SIZE_G(v) ((v) & ((1 << lgsize_g) - 1))
#define DIV_SIZE_G(v) ((v) >> lgsize_g)
#define MUL_SIZE_G(x) ((x) << lgsize_g)

#define VERTEX_OWNER_G(v) ((int)(MOD_SIZE_G(v)))
#define VERTEX_LOCAL_G(v) ((size_t)(DIV_SIZE_G(v)))
#define VERTEX_TO_GLOBAL_G(r, i) ((int64_t)(MUL_SIZE_G((uint64_t)i) + (int)(r)))

// #define LONG_BITS_G (sizeof(unsigned long) * CHAR_BIT)
#define LONG_BITS_G 64

#define SET_LOCAL_G(v, a) do {(a)[VERTEX_LOCAL_G((v)) / LONG_BITS_G] |= (1UL << (VERTEX_LOCAL_G((v)) % LONG_BITS_G));} while (0)
#define TEST_LOCA_G(v, a) (((a)[VERTEX_LOCAL_G((v)) / LONG_BITS_G] & (1UL << (VERTEX_LOCAL_G((v)) % LONG_BITS_G))) != 0)

#define SET_GLOBAL_G(v, a) do {(a)[(v) / LONG_BITS_G] |= (1UL << ((v) % LONG_BITS_G));} while (0)
#define SET_GLOBAL_ATOMIC_G(v, a) do {atomicOr((unsigned long long int*)(&(a)[(v) / LONG_BITS_G]), 1UL << ((v) % LONG_BITS_G));} while (0)
#define TEST_GLOBAL_G(v, a) (((a)[(v) / LONG_BITS_G] & (1UL << ((v) % LONG_BITS_G))) != 0)


extern oned_csr_graph g;
extern int64_t *pred;

extern int64_t *frontier;
extern int64_t *frontier_next;

#define BLOCK_X 64

// do_nothing kernel
// for bottom up
// one thread do one vertex
// one thread try to find parent in its all neighbour
void dim_do_nothing(dim3* dimGrid, dim3 *dimBlock) {
	*dimGrid = dim3((g.nlocalverts + BLOCK_X - 1) / BLOCK_X, 1, 1); // number of block
	*dimBlock = dim3(BLOCK_X, 1, 1); // number of thread per block
}

__global__ void do_nothing(
	int64_t *rowstarts_g, 
	int64_t *column_g, 
	int64_t *frontier_g, 
	int64_t *frontier_next_g, 
	int64_t *pred_g, 
	Context context) {

	set_context(context);

	const int block_base = blockIdx.x * blockDim.x;
	const int64_t i = block_base + threadIdx.x;

    __syncthreads();

	if (i >= nlocalverts_g)
		return ;

    if (pred_g[i] == -1) {
        int j;
        for (j = (int) rowstarts_g[i]; j < rowstarts_g[i + 1]; j++) {
            int64_t parent_global = column_g[j];        
            if (TEST_GLOBAL_G(parent_global, frontier_g)) {
                pred_g[i] = parent_global;
                SET_GLOBAL_ATOMIC_G(VERTEX_TO_GLOBAL_G(rank_g, i), frontier_next_g);
                break;
            }
        }
    }
}

int64_t *rowstarts_g;
int size_rowstarts;
int64_t *column_g;
int size_column;

int64_t *pred_g;
int size_pred_g;

int64_t *frontier_g;
int size_frontier_g;
int64_t *frontier_next_g;
int size_frontier_next_g;


// transfer graph to gpu global memory
// should perform only once
void init_bottom_up_gpu() {
	size_rowstarts = (g.nlocalverts + 1) * sizeof(int64_t);
	size_column = g.rowstarts[g.nlocalverts] * sizeof(int64_t);
	hipMalloc((void **)&rowstarts_g, size_rowstarts);
	hipMalloc((void **)&column_g, size_column);
	hipMemcpy(rowstarts_g, g.rowstarts, size_rowstarts, hipMemcpyHostToDevice);
	hipMemcpy(column_g, g.column, size_column, hipMemcpyHostToDevice);

	// here assume pred always reside in GPU
	// from beginning to end
	// only when everythiing is done
	// transfer pred back to CPU
	size_pred_g = g.nlocalverts * sizeof(int64_t);
	hipMalloc((void **)&pred_g, size_pred_g);
	// hipMemcpy(pred_g, pred, size_pred_g, hipMemcpyHostToDevice);

	size_frontier_g = global_long_nb;
	hipMalloc((void **)&frontier_g, size_frontier_g);
	size_frontier_next_g = global_long_nb;
	hipMalloc((void **)&frontier_next_g, size_frontier_next_g);
}

void pred_to_gpu() {
	hipMemcpy(pred_g, pred, size_pred_g, hipMemcpyHostToDevice);
}

void pred_from_gpu() {
	hipMemcpy(pred, pred_g, size_pred_g, hipMemcpyDeviceToHost);
}

void end_bottom_up_gpu() {
	hipFree(rowstarts_g);
	hipFree(column_g);
	hipFree(pred_g);
	hipFree(frontier_g);
	hipFree(frontier_next_g);
}

void one_step_bottom_up_gpu() {
	// transfer current frontier to gpu
	hipMemcpy(frontier_g, frontier, size_frontier_g, hipMemcpyHostToDevice);
	hipMemset(frontier_next_g, 0, size_frontier_next_g);

	// get suitable dim
	dim3 dimGrid;
	dim3 dimBlock;
	dim_do_nothing(&dimGrid, &dimBlock);

	// launch gpu kernel
	// it should compute frontier_next_g
	Context context = { rank, size, lgsize, g.nlocalverts };
	do_nothing<<<dimGrid, dimBlock>>>(rowstarts_g, column_g, frontier_g, frontier_next_g, pred_g, context);

	hipMemcpy(frontier_next, frontier_next_g, size_frontier_next_g, hipMemcpyDeviceToHost);
}
