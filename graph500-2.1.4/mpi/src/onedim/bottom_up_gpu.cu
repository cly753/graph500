#include "hip/hip_runtime.h"

// 
// CUDA-aware Open MPI
// Now, the Open MPI library will automatically detect that the pointer being passed in is a CUDA device memory pointer and do the right thing. This is referred to as CUDA-aware support.
// https://www.open-mpi.org/faq/?category=runcuda
// https://devblogs.nvidia.com/parallelforall/introduction-cuda-aware-mpi/
// 

extern "C" {
#include "bottom_up.h"
}

#include <string.h>

#include "oned_csr.h"

#include "constants.h"
#include "bfs.h"
#include "print.h"

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h> // helper functions for SDK examples


#ifdef SIZE_MUST_BE_A_POWER_OF_TWO
#define MOD_SIZE_G(v) ((v) & ((1 << lgsize_g) - 1))
#define DIV_SIZE_G(v) ((v) >> lgsize_g)
#define MUL_SIZE_G(x) ((x) << lgsize_g)
#else
#define MOD_SIZE_G(v) ((v) % size_g)
#define DIV_SIZE_G(v) ((v) / size_g)
#define MUL_SIZE_G(x) ((x) * size_g)
#endif
#define VERTEX_OWNER_G(v) ((int)(MOD_SIZE_G(v)))
#define VERTEX_LOCAL_G(v) ((size_t)(DIV_SIZE_G(v)))
#define VERTEX_TO_GLOBAL_G(r, i) ((int64_t)(MUL_SIZE_G((uint64_t)i) + (int)(r)))

// #define LONG_BITS_G (sizeof(unsigned long) * CHAR_BIT)
#define LONG_BITS_G 64

#define SET_LOCAL_G(v, a) do {(a)[VERTEX_LOCAL_G((v)) / LONG_BITS_G] |= (1UL << (VERTEX_LOCAL_G((v)) % LONG_BITS_G));} while (0)
#define TEST_LOCA_G(v, a) (((a)[VERTEX_LOCAL_G((v)) / LONG_BITS_G] & (1UL << (VERTEX_LOCAL_G((v)) % LONG_BITS_G))) != 0)

#define SET_GLOBAL_G(v, a) do {(a)[(v) / LONG_BITS_G] |= (1UL << ((v) % LONG_BITS_G));} while (0)
#define TEST_GLOBAL_G(v, a) (((a)[(v) / LONG_BITS_G] & (1UL << ((v) % LONG_BITS_G))) != 0)

// #define SET_G(v, a) do {(a) |= (1UL << (v));} while (0)
// #define TEST_G(v, a) (((a) & (1UL << (v))) != 0)


extern oned_csr_graph g;
extern int64_t *pred;

extern int64_t *frontier;
extern int64_t *frontier_next;


__device__ int rank_g;
__device__ int size_g;
#ifdef SIZE_MUST_BE_A_POWER_OF_TWO
__device__ int lgsize_g;
#endif
__device__ int64_t nlocalverts_g;


struct Context {
	int rank;
	int size;
#ifdef SIZE_MUST_BE_A_POWER_OF_TWO
	int lgsize;
#endif
	int64_t nlocalverts;
};

__device__ void set_context(Context context) {
	rank_g = context.rank;
	size_g = context.size;
#ifdef SIZE_MUST_BE_A_POWER_OF_TWO
	lgsize_g = context.lgsize;
#endif
	nlocalverts_g = context.nlocalverts;
}



#define BLOCK_X 64

// do_nothing kernel
// for bottom up
// one thread do one vertex
// one thread try to find parent in its all neighbour
void dim_do_nothing(dim3* dimGrid, dim3 *dimBlock) {
	*dimGrid = dim3(128, 1, 1); // number of block
	*dimBlock = dim3(BLOCK_X, 1, 1); // number of thread per block
}

__global__ void do_nothing(
	int64_t *rowstarts_g, 
	int64_t *column_g, 
	int64_t *frontier_g, 
	int64_t *frontier_next_g, 
	int64_t *pred_g, 
	Context context) {

	set_context(context);

	const int block_base = blockIdx.x * blockDim.x;
	const int64_t i = block_base + threadIdx.x;
	__shared__ char next[BLOCK_X];
	next[threadIdx.x] = 0;

    __syncthreads();

	if (i >= nlocalverts_g)
		return ;

    if (pred_g[i] == -1) {
        int j;
        for (j = (int) rowstarts_g[i]; j < rowstarts_g[i + 1]; j++) {
            int64_t parent_global = column_g[j];        
            if (TEST_GLOBAL_G(parent_global, frontier_g)) {
                pred_g[i] = parent_global;
                next[threadIdx.x] = 1;
                break;
            }
        }
    }

    __syncthreads();

    if (threadIdx.x == 0) {
    	int x;
    	for (x = 0; x < BLOCK_X; x++) {
    		if (next[x] == 1) {
	    		int real_i = block_base + x;
	            SET_GLOBAL_G(VERTEX_TO_GLOBAL_G(rank_g, real_i), frontier_next_g);	
    		}
        }
    }
}

int64_t *rowstarts_g;
int size_rowstarts;
int64_t *column_g;
int size_column;

int64_t *pred_g;
int size_pred_g;

int64_t *frontier_g;
int size_frontier_g;
int64_t *frontier_next_g;
int size_frontier_next_g;

// transfer graph to gpu global memory
// should perform only once
void init_bottom_up_gpu() {
	size_rowstarts = (g.nlocalverts + 1) * sizeof(int64_t);
	size_column = g.rowstarts[g.nlocalverts] * sizeof(int64_t);
	hipMalloc((void **)&rowstarts_g, size_rowstarts);
	hipMalloc((void **)&column_g, size_column);
	hipMemcpy(rowstarts_g, g.rowstarts, size_rowstarts, hipMemcpyHostToDevice);
	hipMemcpy(column_g, g.column, size_column, hipMemcpyHostToDevice);

	// here assume pred always reside in GPU
	// from beginning to end
	// only when everythiing is done
	// transfer pred back to CPU
	size_pred_g = g.nlocalverts * sizeof(int64_t);
	hipMalloc((void **)&pred_g, size_pred_g);
	hipMemcpy(pred_g, pred, size_pred_g, hipMemcpyHostToDevice);

	size_frontier_g = global_long_nb;
	hipMalloc((void **)&frontier_g, size_frontier_g);
	size_frontier_next_g = global_long_nb;
	hipMalloc((void **)&frontier_next_g, size_frontier_next_g);
}

void end_bottom_up_gpu() {
	hipMemcpy(pred, pred_g, size_pred_g, hipMemcpyDeviceToHost);

	hipFree(rowstarts_g);
	hipFree(column_g);
	hipFree(pred_g);
	hipFree(frontier_g);
	hipFree(frontier_next_g);
}

// entry to do one level bfs on gpu
// it should transfer graph to gpu global memory
// possibly only once
// and compute suitable grid size, block size
// and launch gpu kernel
// each launch should transfer new frontier
void one_step_bottom_up_gpu() {
	// transfer current frontier and frontier_next to gpu
	hipMemcpy(frontier_g, frontier, size_frontier_g, hipMemcpyHostToDevice);
	hipMemset(frontier_next_g, 0, size_frontier_next_g);
	// hipMemcpy(frontier_next_g, frontier_next, size_frontier_next_g, hipMemcpyHostToDevice);	


	// get suitable dim
	dim3 dimGrid;
	dim3 dimBlock;
	dim_do_nothing(&dimGrid, &dimBlock);

	// launch gpu kernel
	// it should compute frontier_next_g
	Context context = {
		rank, size
#ifdef SIZE_MUST_BE_A_POWER_OF_TWO
		, lgsize
#endif
		, g.nlocalverts
	};

	do_nothing<<<dimGrid, dimBlock>>>(rowstarts_g, column_g, frontier_g, frontier_next_g, pred_g, context);

	hipMemcpy(frontier_next, frontier_next_g, size_frontier_next_g, hipMemcpyDeviceToHost);
}

// 
// nvcc -arch=sm_37 -c bottom_up_gpu.cu -o bottom_up_gpu.o
// 
// GPU
// multiprocessor (SM?): 13
// global memory: 12G
// constant memroy: 64k (2^16)
// shared memory per block: 48k (3 * 2^14)
// register per block: 64k (2^16)
// warp size: 32
// max num thread per multiprocessor (SM?): 2048 (2^11, 2 block)
// max num thread per block: 1024 (2^10)
// 
// 
// x block (max inf)
// y thread per block (max 2^10)
// 
// around 12 int64_t per thread -> 2^7 register -> max 2^9 thread per block
// total 2^14 vertex
// local 2^13 vertex -> 2^13 thread -> at least 2^3 blocks
// 
// at least 2^4 blocks, each 2^9 thread -> 1x SM
// 
// lets try 2^6 blocks=64, each 2^7=128 thread
// 
// frontier bit map: 2^16 / 64 * 8 = 8k
// 
// 
// without considering global memory
// 
// 


// Device 3: "Tesla K80"
//   CUDA Driver Version / Runtime Version          7.5 / 7.5
//   CUDA Capability Major/Minor version number:    3.7
//   Total amount of global memory:                 12288 MBytes (12884705280 bytes)
//   (13) Multiprocessors, (192) CUDA Cores/MP:     2496 CUDA Cores
//   GPU Max Clock rate:                            824 MHz (0.82 GHz)
//   Memory Clock rate:                             2505 Mhz
//   Memory Bus Width:                              384-bit
//   L2 Cache Size:                                 1572864 bytes
//   Maximum Texture Dimension Size (x,y,z)         1D=(65536), 2D=(65536, 65536), 3D=(4096, 4096, 4096)
//   Maximum Layered 1D Texture Size, (num) layers  1D=(16384), 2048 layers
//   Maximum Layered 2D Texture Size, (num) layers  2D=(16384, 16384), 2048 layers
//   Total amount of constant memory:               65536 bytes
//   Total amount of shared memory per block:       49152 bytes
//   Total number of registers available per block: 65536
//   Warp size:                                     32
//   Maximum number of threads per multiprocessor:  2048
//   Maximum number of threads per block:           1024
//   Max dimension size of a thread block (x,y,z): (1024, 1024, 64)
//   Max dimension size of a grid size    (x,y,z): (2147483647, 65535, 65535)
//   Maximum memory pitch:                          2147483647 bytes
//   Texture alignment:                             512 bytes
//   Concurrent copy and kernel execution:          Yes with 2 copy engine(s)
//   Run time limit on kernels:                     No
//   Integrated GPU sharing Host Memory:            No
//   Support host page-locked memory mapping:       Yes
//   Alignment requirement for Surfaces:            Yes
//   Device has ECC support:                        Disabled
//   Device supports Unified Addressing (UVA):      Yes
//   Device PCI Domain ID / Bus ID / location ID:   0 / 133 / 0
//   Compute Mode:
//      < Default (multiple host threads can use ::hipSetDevice() with device simultaneously) >
// > Peer access from Tesla K80 (GPU0) -> Tesla K80 (GPU1) : Yes
// > Peer access from Tesla K80 (GPU0) -> Tesla K80 (GPU2) : No
// > Peer access from Tesla K80 (GPU0) -> Tesla K80 (GPU3) : No
// > Peer access from Tesla K80 (GPU1) -> Tesla K80 (GPU0) : Yes
// > Peer access from Tesla K80 (GPU1) -> Tesla K80 (GPU2) : No
// > Peer access from Tesla K80 (GPU1) -> Tesla K80 (GPU3) : No
// > Peer access from Tesla K80 (GPU2) -> Tesla K80 (GPU0) : No
// > Peer access from Tesla K80 (GPU2) -> Tesla K80 (GPU1) : No
// > Peer access from Tesla K80 (GPU2) -> Tesla K80 (GPU3) : Yes
// > Peer access from Tesla K80 (GPU3) -> Tesla K80 (GPU0) : No
// > Peer access from Tesla K80 (GPU3) -> Tesla K80 (GPU1) : No
// > Peer access from Tesla K80 (GPU3) -> Tesla K80 (GPU2) : Yes
