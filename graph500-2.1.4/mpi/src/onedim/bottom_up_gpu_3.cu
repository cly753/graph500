#include "hip/hip_runtime.h"

// 
// CUDA-aware Open MPI
// Now, the Open MPI library will automatically detect that the pointer being passed in is a CUDA device memory pointer and do the right thing. This is referred to as CUDA-aware support.
// https://www.open-mpi.org/faq/?category=runcuda
// https://devblogs.nvidia.com/parallelforall/introduction-cuda-aware-mpi/
// 

extern "C" {
#include "bottom_up.h"
}

#include <string.h>

#include "oned_csr.h"

#include "constants.h"
#include "bfs.h"
#include "print.h"

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h> // helper functions for SDK examples

// 
// !!! Assume SIZE_MUST_BE_A_POWER_OF_TWO !!!
// 

__device__ int rank_g;
__device__ int size_g;
__device__ int lgsize_g;
__device__ int64_t nlocalverts_g;

struct Context {
	int rank;
	int size;
	int lgsize;
	int64_t nlocalverts;
};

__device__ void set_context(Context context) {
	rank_g = context.rank;
	size_g = context.size;
	lgsize_g = context.lgsize;
	nlocalverts_g = context.nlocalverts;
}

// __device__ int64_t mode_size(int64_t v) {
// 	return v & ((1 << lgsize_g) - 1);
// }

// __device__ int64_t div_size(int64_t v) {
// 	return v >> lgsize_g;
// }

// __device__ int64_t mul_size(int64_t v) {
// 	return v << lgsize_g;
// }

// __device__ int vertex_owner(int64_t v) {
// 	return mode_size(v);
// }

// __device__ size_t vertex_local(int64_t v) {
// 	return div_size(v);
// }

// __device__ int64_t vertex_to_global(int rank, int64_t v) {
// 	return mul_size((uint64_t)v) + rank;
// }

#define LONG_BITS_G 64

// __device__ void set_local(int64_t v, int64_t *a) { // x / 64 --> x >> 6 ??
// 	a[vertex_local(v) / LONG_BITS_G] |= (1UL << (vertex_local(v) % LONG_BITS_G));
// }

// __device__ int test_local(int64_t v, int64_t *a) {
// 	return 0 != (a[vertex_local(v) / LONG_BITS_G] & (1UL << (vertex_local(v) % LONG_BITS_G)));
// }

// __device__ void set_global(int64_t v, int64_t *a) { // x / 64 --> x >> 6 ??
// 	a[v / LONG_BITS_G] |= (1UL << (v % LONG_BITS_G));
// }

// __device__ void set_global_atomic(int64_t v, int64_t *a) {
// 	atomicOr((unsigned long long int*)(&a[v / LONG_BITS_G]), 1UL << (v % LONG_BITS_G));
// }

// __device__ int test_global(int64_t v, int64_t *a) {
// 	return 0 != (a[v / LONG_BITS_G] & (1UL << (v % LONG_BITS_G)));
// }

#define MOD_SIZE_G(v) ((v) & ((1 << lgsize_g) - 1))
#define DIV_SIZE_G(v) ((v) >> lgsize_g)
#define MUL_SIZE_G(x) ((x) << lgsize_g)

#define VERTEX_OWNER_G(v) ((int)(MOD_SIZE_G(v)))
#define VERTEX_LOCAL_G(v) ((size_t)(DIV_SIZE_G(v)))
#define VERTEX_TO_GLOBAL_G(r, i) ((int64_t)(MUL_SIZE_G((uint64_t)i) + (int)(r)))

// #define LONG_BITS_G (sizeof(unsigned long) * CHAR_BIT)
#define LONG_BITS_G 64

#define SET_LOCAL_G(v, a) do {(a)[VERTEX_LOCAL_G((v)) / LONG_BITS_G] |= (1UL << (VERTEX_LOCAL_G((v)) % LONG_BITS_G));} while (0)
#define TEST_LOCA_G(v, a) (((a)[VERTEX_LOCAL_G((v)) / LONG_BITS_G] & (1UL << (VERTEX_LOCAL_G((v)) % LONG_BITS_G))) != 0)

#define SET_GLOBAL_G(v, a) do {(a)[(v) / LONG_BITS_G] |= (1UL << ((v) % LONG_BITS_G));} while (0)
#define SET_GLOBAL_ATOMIC_G(v, a) do {atomicOr((unsigned long long int*)(&(a)[(v) / LONG_BITS_G]), 1UL << ((v) % LONG_BITS_G));} while (0)
#define TEST_GLOBAL_G(v, a) (((a)[(v) / LONG_BITS_G] & (1UL << ((v) % LONG_BITS_G))) != 0)


extern oned_csr_graph g;
extern int64_t *pred;

extern int64_t *frontier;
extern int64_t *frontier_next;

#define BLOCK_X 64

// do_nothing kernel
// for bottom up
// one thread do one vertex
// one thread try to find parent in its all neighbour
void dim_do_nothing(dim3* dimGrid, dim3 *dimBlock) {
	*dimGrid = dim3((g.rowstarts[g.nlocalverts] + BLOCK_X - 1) / BLOCK_X, 1, 1); // number of block
	*dimBlock = dim3(BLOCK_X, 1, 1); // number of thread per block
}

__global__ void do_nothing(
	int64_t *row_g, 
	int64_t *column_g, 
	int64_t *frontier_g, 
	int64_t *frontier_next_g, 
	int64_t *pred_g,
	int64_t total_edge,
	Context context) {

	set_context(context);

	const int block_base = blockIdx.x * blockDim.x;
	const int i = block_base + threadIdx.x;

	if (i >= total_edge)
		return ;

	int from = row_g[i];
	if (pred_g[from] == -1) {
		int to_global = column_g[i];
		if (TEST_GLOBAL_G(to_global, frontier_g)) {
			pred_g[from] = to_global;
			// pred_g[from] = 999;
			SET_GLOBAL_ATOMIC_G(VERTEX_TO_GLOBAL_G(rank_g, from), frontier_next_g);
		}
	}
}

int64_t *rowstarts_g;
int size_rowstarts;
int64_t *column_g;
int size_column;

int64_t *row_g;
int size_row;

int64_t *pred_g;
int size_pred_g;

int64_t *frontier_g;
int size_frontier_g;
int64_t *frontier_next_g;
int size_frontier_next_g;


__global__ void fill_row_g(int64_t *rowstarts_g, int64_t *row_g, int nlocalverts) {
	const int block_base = blockIdx.x * blockDim.x;
	const int i = block_base + threadIdx.x;
	if (i >= nlocalverts)
		return ;
	int j;
	for (j = rowstarts_g[i]; j < rowstarts_g[i + 1]; j++) {
		row_g[j] = i;
	}
}

// transfer graph to gpu global memory
// should perform only once
void init_bottom_up_gpu() {
	size_rowstarts = (g.nlocalverts + 1) * sizeof(int64_t);
	size_column = g.rowstarts[g.nlocalverts] * sizeof(int64_t);
	hipMalloc((void **)&rowstarts_g, size_rowstarts);
	hipMalloc((void **)&column_g, size_column);
	hipMemcpy(rowstarts_g, g.rowstarts, size_rowstarts, hipMemcpyHostToDevice);
	hipMemcpy(column_g, g.column, size_column, hipMemcpyHostToDevice);

	size_row = size_column;
	hipMalloc((void **)&row_g, size_row);
	fill_row_g<<<(g.nlocalverts + BLOCK_X - 1) / BLOCK_X, BLOCK_X>>>(rowstarts_g, row_g, g.nlocalverts);

	// here assume pred always reside in GPU
	// from beginning to end
	// only when everythiing is done
	// transfer pred back to CPU
	size_pred_g = g.nlocalverts * sizeof(int64_t);
	hipMalloc((void **)&pred_g, size_pred_g);
	// hipMemcpy(pred_g, pred, size_pred_g, hipMemcpyHostToDevice);

	size_frontier_g = global_long_nb;
	hipMalloc((void **)&frontier_g, size_frontier_g);
	size_frontier_next_g = global_long_nb;
	hipMalloc((void **)&frontier_next_g, size_frontier_next_g);
}

void pred_to_gpu() {
	hipMemcpy(pred_g, pred, size_pred_g, hipMemcpyHostToDevice);
}

void pred_from_gpu() {
	hipMemcpy(pred, pred_g, size_pred_g, hipMemcpyDeviceToHost);
}

void end_bottom_up_gpu() {
	hipFree(rowstarts_g);
	hipFree(column_g);
	hipFree(pred_g);
	hipFree(frontier_g);
	hipFree(frontier_next_g);

	hipFree(row_g);
}

void one_step_bottom_up_gpu() {
	// transfer current frontier to gpu
	hipMemcpy(frontier_g, frontier, size_frontier_g, hipMemcpyHostToDevice);
	hipMemset(frontier_next_g, 0, size_frontier_next_g);

	// get suitable dim
	dim3 dimGrid;
	dim3 dimBlock;
	dim_do_nothing(&dimGrid, &dimBlock);

	// launch gpu kernel
	// it should compute frontier_next_g
	Context context = { rank, size, lgsize, g.nlocalverts };
	do_nothing<<<dimGrid, dimBlock>>>(row_g, column_g, frontier_g, frontier_next_g, pred_g, g.rowstarts[g.nlocalverts], context);

	hipMemcpy(frontier_next, frontier_next_g, size_frontier_next_g, hipMemcpyDeviceToHost);
}
