#include "hip/hip_runtime.h"

// 
// CUDA-aware Open MPI
// Now, the Open MPI library will automatically detect that the pointer being passed in is a CUDA device memory pointer and do the right thing. This is referred to as CUDA-aware support.
// https://www.open-mpi.org/faq/?category=runcuda
// https://devblogs.nvidia.com/parallelforall/introduction-cuda-aware-mpi/
// 

extern "C" {
#include "bottom_up.h"
}

#include <string.h>

#include "oned_csr.h"

#include "constants.h"
#include "bfs.h"
#include "frontier_tracker.h"
#include "print.h"

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h> // helper functions for SDK examples

// 
// !!! Assume SIZE_MUST_BE_A_POWER_OF_TWO !!!
// 

__device__ int rank_g;
__device__ int size_g;
__device__ int lgsize_g;
__device__ int64_t nlocalverts_g;

struct Context {
	int rank;
	int size;
	int lgsize;
	int64_t nlocalverts;
};

__device__ void set_context(Context context) {
	rank_g = context.rank;
	size_g = context.size;
	lgsize_g = context.lgsize;
	nlocalverts_g = context.nlocalverts;
}

// __device__ int64_t mode_size(int64_t v) {
// 	return v & ((1 << lgsize_g) - 1);
// }

// __device__ int64_t div_size(int64_t v) {
// 	return v >> lgsize_g;
// }

// __device__ int64_t mul_size(int64_t v) {
// 	return v << lgsize_g;
// }

// __device__ int vertex_owner(int64_t v) {
// 	return mode_size(v);
// }

// __device__ size_t vertex_local(int64_t v) {
// 	return div_size(v);
// }

// __device__ int64_t vertex_to_global(int rank, int64_t v) {
// 	return mul_size((uint64_t)v) + rank;
// }

// __device__ void set_local(int64_t v, int64_t *a) { // x / 64 --> x >> 6 ??
// 	a[vertex_local(v) / LONG_BITS_G] |= (1UL << (vertex_local(v) % LONG_BITS_G));
// }

// __device__ int test_local(int64_t v, int64_t *a) {
// 	return 0 != (a[vertex_local(v) / LONG_BITS_G] & (1UL << (vertex_local(v) % LONG_BITS_G)));
// }

// __device__ void set_global(int64_t v, int64_t *a) { // x / 64 --> x >> 6 ??
// 	a[v / LONG_BITS_G] |= (1UL << (v % LONG_BITS_G));
// }

// __device__ void set_global_atomic(int64_t v, int64_t *a) {
// 	atomicOr((unsigned long long int*)(&a[v / LONG_BITS_G]), 1UL << (v % LONG_BITS_G));
// }

// __device__ int test_global(int64_t v, int64_t *a) {
// 	return 0 != (a[v / LONG_BITS_G] & (1UL << (v % LONG_BITS_G)));
// }


extern oned_csr_graph g;
extern int64_t *pred;

extern int64_t *frontier;
extern int64_t *frontier_next;

#define BLOCK_X 64

// do_nothing kernel
// for bottom up
// one thread do one vertex
// one thread try to find parent in its all neighbour

// get dim for bfs kernel
void dim_do_nothing(dim3* dimGrid, dim3 *dimBlock) {
	*dimGrid = dim3((g.rowstarts[g.nlocalverts] + BLOCK_X - 1) / BLOCK_X, 1, 1); // number of block
	*dimBlock = dim3(BLOCK_X, 1, 1); // number of thread per block
}

// bfs kernel
// each thread do one edge
__global__ void do_nothing(
	int64_t *row_g, 
	int64_t *column_g, 
	int64_t *frontier_g, 
	int64_t *frontier_next_g, 
	int64_t *pred_g,
	int64_t total_edge,
	Context context) {

	set_context(context);

	const int block_base = blockIdx.x * blockDim.x;
	const int i = block_base + threadIdx.x; // this thread do i-th edge

	if (i >= total_edge)
		return ;

	int from = row_g[i]; // one end of the edge
	if (pred_g[from] == -1) { // bottom up, so check if from is unvisited
		int to_global = column_g[i]; // the other end of the edge
		if (TEST_GLOBAL_G(to_global, frontier_g)) { // check if is in frontier
			pred_g[from] = to_global;
			SET_GLOBAL_ATOMIC_G(VERTEX_TO_GLOBAL_G(rank_g, from), frontier_next_g);
		}
	}
}

int64_t *rowstarts_g;
int size_rowstarts;
int64_t *column_g;
int size_column;

int64_t *row_g;
int size_row;

int64_t *pred_g;
int size_pred_g;

int64_t *frontier_g;
int size_frontier_g;
int64_t *frontier_next_g;
int size_frontier_next_g;

void show_pred_g() {
	int64_t *pred_copy = (int64_t *)xmalloc(size_pred_g);
	hipMemcpy(pred_copy, pred_g, size_pred_g, hipMemcpyDeviceToHost);

    PRINT_RANK("gpu index:")
    for (int i = 0; i < g.nlocalverts; i++) {
        PRINT(" %"PRId64"", (i * size + rank))
    }
    PRINTLN("")
    PRINT_RANK("gpu pred :")
    for (int i = 0; i < g.nlocalverts; i++) {
        PRINT(" %"PRId64"", pred_copy[i])
    }
    PRINTLN("")

    free(pred_copy);
}

void read_frontier_next_g() {
	hipMemcpy(frontier_next, frontier_next_g, global_long_nb, hipMemcpyDeviceToHost);
}

void save_frontier_g() {
	hipMemcpy(frontier_g, frontier, global_long_nb, hipMemcpyHostToDevice);
}

int64_t* get_frontier_g() {
	int64_t *frontier_g_copy = (int64_t *)xmalloc(global_long_nb);
	hipMemcpy(frontier_g_copy, frontier_g, global_long_nb, hipMemcpyDeviceToHost);	
	return frontier_g_copy;
}

int64_t* get_frontier_next_g() {
	int64_t *frontier_next_g_copy = (int64_t *)xmalloc(global_long_nb);
	hipMemcpy(frontier_next_g_copy, frontier_next_g, global_long_nb, hipMemcpyDeviceToHost);
	return frontier_next_g_copy;
}

// use CPU to fill?
__global__ void fill_row_g(int64_t *rowstarts_g, int64_t *row_g, int nlocalverts) {
	const int block_base = blockIdx.x * blockDim.x;
	const int i = block_base + threadIdx.x;
	if (i >= nlocalverts)
		return ;
	for (int j = rowstarts_g[i]; j < rowstarts_g[i + 1]; j++)
		row_g[j] = i;
}

__global__ void fill_row_g_binary(int64_t *rowstarts_g, int64_t *row_g, int nlocalverts, int64_t total_edge) {
	const int block_base = blockIdx.x * blockDim.x;
	const int i = block_base + threadIdx.x;
	if (i >= total_edge)
		return ;
	int l = 0;
	int r = nlocalverts;
	// int r = i;
	while (1) {
		int m = (l + r) / 2;
		int a = rowstarts_g[m];
		int b = rowstarts_g[m + 1];
		if (a <= i && i < b) {
			row_g[i] = m;
			break;
		}
		else if (b <= i) {
			l = m + 1;
		}
		else {
			r = m;
		}
	}
}

// transfer graph to gpu global memory
// should perform only once
void init_bottom_up_gpu() {
	size_rowstarts = (g.nlocalverts + 1) * sizeof(int64_t);
	size_column = g.rowstarts[g.nlocalverts] * sizeof(int64_t);
	hipMalloc((void **)&rowstarts_g, size_rowstarts);
	hipMalloc((void **)&column_g, size_column);
	hipMemcpy(rowstarts_g, g.rowstarts, size_rowstarts, hipMemcpyHostToDevice);
	hipMemcpy(column_g, g.column, size_column, hipMemcpyHostToDevice);

	size_row = size_column;
	hipMalloc((void **)&row_g, size_row);
	fill_row_g<<<(g.nlocalverts + BLOCK_X - 1) / BLOCK_X, BLOCK_X>>>(rowstarts_g, row_g, g.nlocalverts);
	// fill_row_g_binary<<<(g.rowstarts[g.nlocalverts] + BLOCK_X - 1) / BLOCK_X, BLOCK_X>>>(rowstarts_g, row_g, g.nlocalverts, g.rowstarts[g.nlocalverts]);

	// here assume pred always reside in GPU
	// from beginning to end
	// only when everythiing is done
	// transfer pred back to CPU
	size_pred_g = g.nlocalverts * sizeof(int64_t);
	hipMalloc((void **)&pred_g, size_pred_g);
	// hipMemcpy(pred_g, pred, size_pred_g, hipMemcpyHostToDevice);

	size_frontier_g = global_long_nb;
	hipMalloc((void **)&frontier_g, size_frontier_g);
	hipMemset(frontier_g, 0, size_frontier_g);
	size_frontier_next_g = global_long_nb;
	hipMalloc((void **)&frontier_next_g, size_frontier_next_g);
}

// no need to use if cuda ompi
void pred_to_gpu() {
	hipMemcpy(pred_g, pred, size_pred_g, hipMemcpyHostToDevice);
}

// use this if cuda ompi
void init_pred_gpu(int64_t root, int is_root_owner) {
	hipMemset(pred_g, -1, size_pred_g);
	if (is_root_owner) {
		// http://stackoverflow.com/questions/7464015/cuda-change-single-value-in-array
		hipMemcpy(pred_g + VERTEX_LOCAL(root), &root, sizeof(int64_t), hipMemcpyHostToDevice); 
	}
#ifdef SHOWDEBUG
	show_pred_g();
#endif
}

void pred_from_gpu() {
	hipMemcpy(pred, pred_g, size_pred_g, hipMemcpyDeviceToHost);
}

void end_bottom_up_gpu() {
	hipFree(rowstarts_g);
	hipFree(column_g);
	hipFree(pred_g);
	hipFree(frontier_g);
	hipFree(frontier_next_g);

	hipFree(row_g);
}

void one_step_bottom_up_gpu() {
	hipMemset(frontier_next_g, 0, size_frontier_next_g);

	// get suitable dim
	dim3 dimGrid;
	dim3 dimBlock;
	dim_do_nothing(&dimGrid, &dimBlock);

	// launch gpu kernel
	// it should compute frontier_next_g
	Context context = { rank, size, lgsize, g.nlocalverts };
	do_nothing<<<dimGrid, dimBlock>>>(row_g, column_g, frontier_g, frontier_next_g, pred_g, g.rowstarts[g.nlocalverts], context);
#ifdef SHOWDEBUG
	show_pred_g();
#endif
}

void set_frontier_gpu(int64_t v) {
	// http://stackoverflow.com/questions/7464015/cuda-change-single-value-in-array
	int the_long = v / LONG_BITS_G;
	int64_t val = 1UL << (v % LONG_BITS_G);
	hipMemcpy(frontier_g + the_long, &val, sizeof(int64_t), hipMemcpyHostToDevice); 
}

__device__ int have_more_g;

__global__ void check_have_more(int64_t *frontier_g, int global_long_n) {
	const int block_base = blockIdx.x * blockDim.x;
	const int i = block_base + threadIdx.x;
	if (i >= global_long_n)
		return ;

	__shared__ int hm;
	hm = 0;
	__syncthreads();
	if (frontier_g[i])
		hm = 1;
	__syncthreads();
	if (threadIdx.x == 0 && hm)
		have_more_g = 1;
}

// __global__ void reset_have_more_g() {
// 	have_more_g = 0;
// }

#define HAVE_MORE_BLOCK_SIZE 64

// gpu version of checking if frontier is empty
// so check if all frontier int64_t is 0 or not
int frontier_have_more_gpu() {
	int have_more = 0;

	int grid_size = (global_long_n + HAVE_MORE_BLOCK_SIZE - 1) / HAVE_MORE_BLOCK_SIZE;
	// reset_have_more_g<<<1, 1>>>();
	hipMemcpyToSymbol(HIP_SYMBOL(have_more_g), &have_more, sizeof(int), 0, hipMemcpyHostToDevice);
	check_have_more<<<grid_size, HAVE_MORE_BLOCK_SIZE>>>(frontier_g, global_long_n);

	hipMemcpyFromSymbol(&have_more, HIP_SYMBOL(have_more_g), sizeof(int), 0, hipMemcpyDeviceToHost);
	return have_more;
}
